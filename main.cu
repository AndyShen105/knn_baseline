#include "hip/hip_runtime.h"
#include <memory>
#include <iostream>
#include <vector>
#include "data.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
//temp
#include <unordered_map>
#include <regex>
#include <queue>
#include "util.h"
#include "data.h"
#include "lsh.h"
#include "v_lsh.h"
#include "hashFunc.h"
#include<cmath>

using namespace std;
#define CLOCKS_PER_SECOND 1000000.0
#define DEBUG 0
__global__ void matrixMulCosine(float* d_data, float * d_query, float * d_result, int *uIndex, int *qIndex, int q, int n, int n_feats);
__global__ void preProcess(float* d_data, float * d_query, float * d_Udot, float * d_Sdot, int n, int q, int n_feats);
__global__ void matrixMulCosineOpt(float* d_data, float * d_query, float * d_result, int *uIndex, int *qIndex, int q, int n,
                                   float* d_Udot, float* d_Sdot,int n_feats);
int* vector2int(vector<int> v){
    int* pV;
    int size = v.size();
    pV = (int *)malloc(sizeof(int)*size);
    for(int i=0; i<size; i++){
        pV[i] = v[i];
    }
    return pV;
}

void gen_ExAudiences_cuda(priority_queue<canducate_user> &top_k,
                          unordered_map<int,vector<int>> user_maps_pool,
                          unordered_map<int,vector<int>> user_maps_seed,
                          priority_queue<uncertain_user> &user_pool,
                          vector<bucket_info> centroid_angle,
                          int n_bit,
                          int n_feats,
                          int k,
                          int n,
                          int q,
                          float * data,
                          float *queries){

    int n_cycle = pow(2, n_bit);
    long long all_count = 0;
    long long save_calu_times = 0;
    long long sum_save_times = 0;
    long query_Bytes = sizeof(float)*q*n_feats;
    long data_Bytes = sizeof(float)*n*n_feats;


    // 申请数据device内存
    float  *d_data, *d_query;
    hipMalloc((void**)&d_query, query_Bytes);
    hipMalloc((void**)&d_data, data_Bytes);
    // copy数据到device
    hipMemcpy((void*)d_query, (void*) queries, query_Bytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_data, (void*) data, data_Bytes, hipMemcpyHostToDevice);

    canducate_user temp_user;
    uncertain_user user;

    for(int i=0; i<n_cycle; i++){

        if (user_maps_seed[i].empty()){
            continue;
        }
        vector<int> &seed = user_maps_seed[i];
        vector<int> &pool = user_maps_pool[i];

        // if size<10, lsh, else v-lsh
        if (seed.size()<10){
            for(vector<int>::const_iterator pool_index=pool.cbegin(); pool_index!=pool.cend(); pool_index++){
                sum_save_times+=seed.size();
                temp_user = calculate_similarity(seed, *pool_index, n_feats, data, queries);
                if (top_k.size() == k && temp_user.sim > top_k.top().sim )
                    top_k.pop();
                if (top_k.size() < k )
                    top_k.push(temp_user);
            }
        }
        else{

            int *uIndex, *qIndex;
            uIndex = (int *)malloc(sizeof(int)*pool.size());
            int uCount = 0;
            for(vector<int>::const_iterator pool_index=pool.cbegin(); pool_index!=pool.cend(); pool_index++){
                float upper_bound = calculate_upperbound_per_user(*pool_index, data, centroid_angle[i].centroid,
                                                                  centroid_angle[i].centroid_sqrt,
                                                                  centroid_angle[i].theta_b, n_feats);
                if (upper_bound == 1.0) {
                    uIndex[uCount] = *pool_index;
                    uCount++;
                } else {
                    user.bucket_no = i;
                    user.index = *pool_index;
                    user.upperbound = upper_bound;
                    user_pool.push(user);
                }
            }
            //申请索引内存
            float *result, *d_result;
            int *d_qIndex, *d_uIndex;
            long result_Bytes = sizeof(float)*seed.size()*uCount;
            long qIndex_Bytes = sizeof(int)*seed.size();
            long uIndex_Bytes = sizeof(int)*pool.size();
            result = (float *)malloc(result_Bytes);
            hipMalloc((void**)&d_result, result_Bytes);
            hipMalloc((void**)&d_qIndex, qIndex_Bytes);
            hipMalloc((void**)&d_uIndex, uIndex_Bytes);

            //copy索引数据到device
            qIndex = vector2int(seed);
            hipMemcpy((void*)d_qIndex, (void*) qIndex, qIndex_Bytes, hipMemcpyHostToDevice);
            hipMemcpy((void*)d_uIndex, (void*) uIndex, uIndex_Bytes, hipMemcpyHostToDevice);

            dim3 blockSize(1024);
            dim3 gridSize((uCount*seed.size() + blockSize.x - 1) / blockSize.x);

            cout<<"uCount: "<<uCount<<" qCount: "<<seed.size()<<endl;
            cout<<"sum thread: "<<uCount*seed.size()<<"grid: "<<gridSize.x<<" block:"<<blockSize.x<<endl;

            matrixMulCosine<< < gridSize, blockSize >> >(d_data, d_query, d_result, d_uIndex, d_qIndex, seed.size(), uCount, n_feats);
            hipDeviceSynchronize();
            hipMemcpy((void*)result, (void*)d_result, result_Bytes, hipMemcpyDeviceToHost);
            for(int i=0; i<uCount; i++){
                float tempSim = 0.0;
                for(int j=0; j<seed.size(); j++){
                    tempSim = max(tempSim, result[i*seed.size()+j]);
                }
                temp_user.sn = uIndex[i];
                temp_user.sim = tempSim;
                if (top_k.size() == k && temp_user.sim > top_k.top().sim && temp_user.sim != -1000.0)
                    top_k.pop();
                if (top_k.size() < k && temp_user.sim != -1000.0)
                    top_k.push(temp_user);

            }
            //释放申请的内存
            hipFree(d_result);
            hipFree(d_qIndex);
            hipFree(d_uIndex);
            free(result);
            free(qIndex);
            free(uIndex);


        }
    }
    hipFree(d_data);
    hipFree(d_query);
    cout<<"size of candicate user with uppbound"<<user_pool.size()<<endl;
    while(!user_pool.empty()){
        user = user_pool.top();
        user_pool.pop();
        int bucket_no = user.bucket_no;
        int index = user.index;
        float upperbound = user.upperbound;
        if( upperbound >= top_k.top().sim ){
            vector<int> &seed = user_maps_seed[bucket_no];
            temp_user = calculate_similarity(seed, index, n_feats, data, queries);
            if (top_k.size() == k && temp_user.sim > top_k.top().sim && temp_user.sim != -1000.0)
                top_k.pop();
            if (top_k.size() < k && temp_user.sim != -1000.0)
                top_k.push(temp_user);
        }else{
            all_count++;
            save_calu_times += user_maps_seed[bucket_no].size();
        }
    }
    cout<<"all_count: "<<all_count<<endl;
    cout<<"save times: "<< save_calu_times<<endl;

}

void gen_ExAudiences_cudaOpt(priority_queue<canducate_user> &top_k,
                          unordered_map<int,vector<int>> user_maps_pool,
                          unordered_map<int,vector<int>> user_maps_seed,
                          priority_queue<uncertain_user> &user_pool,
                          vector<bucket_info> centroid_angle,
                          int n_bit,
                          int n_feats,
                          int k,
                          int n,
                          int q,
                          float * data,
                          float *queries){

    int n_cycle = pow(2, n_bit);
    long long all_count = 0;
    long long save_calu_times = 0;
    long long sum_save_times = 0;
    long query_Bytes = sizeof(float)*q*n_feats;
    long data_Bytes = sizeof(float)*n*n_feats;


    // 申请数据device内存
    float  *d_data, *d_query, *d_Udot, *d_Sdot;
    hipMalloc((void**)&d_query, query_Bytes);
    hipMalloc((void**)&d_data, data_Bytes);
    hipMalloc((void**)&d_Udot, sizeof(float)*n);
    hipMalloc((void**)&d_Sdot, sizeof(float)*q);
    // copy数据到device
    hipMemcpy((void*)d_query, (void*) queries, query_Bytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_data, (void*) data, data_Bytes, hipMemcpyHostToDevice);

    dim3 preProblockSize(1024);
    dim3 preProgridSize((n+q+preProblockSize.x - 1) / preProblockSize.x);

    preProcess<< < preProgridSize, preProblockSize >> >(d_data, d_query, d_Udot, d_Sdot, n, q, n_feats);

    cout<<"grid: "<<preProgridSize.x<<" block:"<<preProblockSize.x<<endl;

    canducate_user temp_user;
    uncertain_user user;

    for(int i=0; i<n_cycle; i++){

        if (user_maps_seed[i].empty()){
            continue;
        }
        vector<int> &seed = user_maps_seed[i];
        vector<int> &pool = user_maps_pool[i];

        // if size<10, lsh, else v-lsh
        if (seed.size()<10){
            for(vector<int>::const_iterator pool_index=pool.cbegin(); pool_index!=pool.cend(); pool_index++){
                sum_save_times+=seed.size();
                temp_user = calculate_similarity(seed, *pool_index, n_feats, data, queries);
                if (top_k.size() == k && temp_user.sim > top_k.top().sim )
                    top_k.pop();
                if (top_k.size() < k )
                    top_k.push(temp_user);
            }
        }
        else{

            int *uIndex, *qIndex;
            uIndex = (int *)malloc(sizeof(int)*pool.size());
            int uCount = 0;
            for(vector<int>::const_iterator pool_index=pool.cbegin(); pool_index!=pool.cend(); pool_index++){
                float upper_bound = calculate_upperbound_per_user(*pool_index, data, centroid_angle[i].centroid,
                                                                  centroid_angle[i].centroid_sqrt,
                                                                  centroid_angle[i].theta_b, n_feats);
                if (upper_bound == 1.0) {
                    uIndex[uCount] = *pool_index;
                    uCount++;
                } else {
                    user.bucket_no = i;
                    user.index = *pool_index;
                    user.upperbound = upper_bound;
                    user_pool.push(user);
                }
            }
            //申请索引内存
            float *result, *d_result;
            int *d_qIndex, *d_uIndex;
            long result_Bytes = sizeof(float)*seed.size()*uCount;
            long qIndex_Bytes = sizeof(int)*seed.size();
            long uIndex_Bytes = sizeof(int)*pool.size();
            result = (float *)malloc(result_Bytes);
            hipMalloc((void**)&d_result, result_Bytes);
            hipMalloc((void**)&d_qIndex, qIndex_Bytes);
            hipMalloc((void**)&d_uIndex, uIndex_Bytes);

            //copy索引数据到device
            qIndex = vector2int(seed);
            hipMemcpy((void*)d_qIndex, (void*) qIndex, qIndex_Bytes, hipMemcpyHostToDevice);
            hipMemcpy((void*)d_uIndex, (void*) uIndex, uIndex_Bytes, hipMemcpyHostToDevice);

            dim3 blockSize(1024);
            dim3 gridSize((uCount*seed.size() + blockSize.x - 1) / blockSize.x);

//            cout<<"uCount: "<<uCount<<" qCount: "<<seed.size()<<endl;
//            cout<<"sum thread: "<<uCount*seed.size()<<"grid: "<<gridSize.x<<" block:"<<blockSize.x<<endl;

            matrixMulCosineOpt<< < gridSize, blockSize >> >(d_data, d_query,d_result, uIndex,qIndex,  seed.size(), uCount, d_Udot, d_Sdot, n_feats);
            hipDeviceSynchronize();
            hipMemcpy((void*)result, (void*)d_result, result_Bytes, hipMemcpyDeviceToHost);
            for(int i=0; i<uCount; i++){
                float tempSim = 0.0;
                for(int j=0; j<seed.size(); j++){
                    tempSim = max(tempSim, result[i*seed.size()+j]);
                }
                temp_user.sn = uIndex[i];
                temp_user.sim = tempSim;
                if (top_k.size() == k && temp_user.sim > top_k.top().sim && temp_user.sim != -1000.0)
                    top_k.pop();
                if (top_k.size() < k && temp_user.sim != -1000.0)
                    top_k.push(temp_user);

            }
            //释放申请的内存
            hipFree(d_result);
            hipFree(d_qIndex);
            hipFree(d_uIndex);
            hipFree(d_Udot);
            hipFree(d_Sdot);
            free(result);
            free(qIndex);
            free(uIndex);


        }
    }
    hipFree(d_data);
    hipFree(d_query);
    cout<<"size of candicate user with uppbound"<<user_pool.size()<<endl;
    while(!user_pool.empty()){
        user = user_pool.top();
        user_pool.pop();
        int bucket_no = user.bucket_no;
        int index = user.index;
        float upperbound = user.upperbound;
        if( upperbound >= top_k.top().sim ){
            vector<int> &seed = user_maps_seed[bucket_no];
            temp_user = calculate_similarity(seed, index, n_feats, data, queries);
            if (top_k.size() == k && temp_user.sim > top_k.top().sim && temp_user.sim != -1000.0)
                top_k.pop();
            if (top_k.size() < k && temp_user.sim != -1000.0)
                top_k.push(temp_user);
        }else{
            all_count++;
            save_calu_times += user_maps_seed[bucket_no].size();
        }
    }
    cout<<"all_count: "<<all_count<<endl;
    cout<<"save times: "<< save_calu_times<<endl;

}


void gen_ExAudiences (int n_feats, int q, int n, float * data, float *queries){

    long query_Bytes = sizeof(float)*q*n_feats;
    long data_Bytes = sizeof(float)*n*n_feats;
    long result_Bytes = sizeof(float)*q*n;

    // 申请device内存
    float *result, *d_data, *d_query, *d_result;
    result = (float *)malloc(result_Bytes);
    hipMalloc((void**)&d_query, query_Bytes);
    hipMalloc((void**)&d_data, data_Bytes);
    hipMalloc((void**)&d_result, result_Bytes);

    clock_t time1 = clock();
    hipMemcpy((void*)d_query, (void*) queries, query_Bytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_data, (void*) data, data_Bytes, hipMemcpyHostToDevice);
    clock_t time2 = clock();

    dim3 blockSize(1024);
    cout<<n*q<<endl;
    dim3 gridSize((n*q + blockSize.x - 1) / blockSize.x);

    cout<<blockSize.x<<endl;
    cout<<gridSize.x<<endl;

    cout<<"tranfer data time: "<<(time2-time1)/CLOCKS_PER_SECOND<<endl;

    //matrixMulCosine << < gridSize, blockSize >> >( d_data, d_query, d_result, q, n, n_feats);
    hipMemcpy((void*)result, (void*)d_result, result_Bytes, hipMemcpyDeviceToHost);
    clock_t time3 = clock();
    cout<<"matrixmul time: "<<(time3-time2)/CLOCKS_PER_SECOND<<endl;
    for(int i=0; i<50; i++){
        cout<<result[i]<<endl;
    }
    hipFree(d_data);
    hipFree(d_query);
    hipFree(d_result);
    free(result);
    // 释放host内存

}

__device__ float dotProduct(float* x, float * y, int x_index, int y_index, int n_feats){
    float temp = 0.0;
    for(int i=0; i<n_feats; i++){
        temp += x[x_index+i]*y[y_index+i];
    }
    return temp;
}


__global__ void matrixMulCosine(float* d_data, float * d_query, float * d_result, int *uIndex, int *qIndex, int q, int n, int n_feats)
{

    // 获取索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int n_index = uIndex[index/q]*n_feats;
    int q_index = qIndex[index%q]*n_feats;
    //printf("index: %d n_index: %d q_index: %d \r\n", index, n_index/50, q_index/50);
    //关闭多余的线程
    if(index>=n*q)
        return;

    //初始化结果为0
    d_result[index]=0.0;
    float x,y,dot,temp;
    x = dotProduct(d_query, d_query, q_index, q_index, n_feats);
    y = dotProduct(d_data, d_data, n_index, n_index, n_feats);
    dot = dotProduct(d_data, d_query, n_index, q_index, n_feats);
    temp = dot/(sqrt(x)*sqrt(y));
/*
    printf("n_index: %d \r\n",n_index/50);
    printf("d_index: %d \r\n",q_index/50);
    if(index/q==0)
        printf("index: %d  temp: %f \r\n",index/q, temp);
 */
    d_result[index] = max(d_result[index], temp);

}
/*----------------------------------------------------------------*/
__global__ void preProcess(float* d_data, float * d_query, float * d_Udot, float * d_Sdot, int n, int q, int n_feats){
    int index,n_index;
    index = threadIdx.x + blockIdx.x * blockDim.x;
    n_index = index-q;
    printf("q: %d \r\n ",q);
    printf("n_index: %d index: %d \r\n ",n_index, index);
    if(n_index>n)
        return;

    float x;
    if(n_index<0){
        x = dotProduct(d_query, d_query, index*n_feats, index*n_feats, n_feats);
        d_Sdot[index]=sqrt(x);

r
    }else{
        x = dotProduct(d_data, d_data, n_index*n_feats, n_index*n_feats, n_feats);
        d_Udot[n_index]=sqrt(x);
        //printf("d_Udot: %d index: %d n_index: %d q: %d \r\n ",sqrt(x), index, n_index, q);

    }
}

__global__ void matrixMulCosineOpt(float* d_data, float * d_query, float * d_result, int *uIndex, int *qIndex, int q, int n,
                                   float* d_Udot, float* d_Sdot,int n_feats)
{

    // 获取索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int n_index = uIndex[index/q]*n_feats;
    int q_index = qIndex[index%q]*n_feats;
    //printf("index: %d n_index: %d q_index: %d \r\n", index, n_index/50, q_index/50);
    //关闭多余的线程
    if(index>=n*q)
        return;

    //初始化结果为0
    d_result[index]=0.0;
    float dot,temp;
    dot = dotProduct(d_data, d_query, n_index, q_index, n_feats);
    temp = dot/(sqrt(d_Sdot[qIndex[index%q]])*sqrt(d_Udot[uIndex[index/q]]));
/*
    printf("n_index: %d \r\n",n_index/50);
    printf("d_index: %d \r\n",q_index/50);
    if(index/q==0)
        printf("index: %d  temp: %f \r\n",index/q, temp);
 */
    d_result[index] = max(d_result[index], temp);

}
/*----------------------------------------------------------------*/
#if 0
int main()
{
    float *queries, *data;
    data = (float *)malloc((int64_t)sizeof(float)*50*50000);
    csv_to_array(&data, "/home/andyshen/data/MovieLens/q_50000.txt", 50000, 50);
    queries = (float *)malloc((int64_t)sizeof(float)*50*10000);
    csv_to_array(&queries, "/home/andyshen/data/MovieLens/p_10000.txt", 10000, 50);
    gen_ExAudiences (50, 10000, 50000, data, queries);

    free(data);
    free(queries);

    return 0;
}
#endif
int main(){
    float *data,*queries;
    data = (float *)malloc((int64_t)sizeof(float)*50*247753);
    csv_to_array(&data, "/home/andyshen/data/MovieLens/q.txt", 247753, 50);
    queries = (float *)malloc((int64_t)sizeof(float)*50*33670);
    csv_to_array(&queries, "/home/andyshen/data/MovieLens/p.txt", 33670, 50);
    cout<<"read data done"<<endl;
    clock_t time1 = clock();
    float** sig_maritx = gen_signature_matrix(50, 5);
    clock_t time2 = clock();
    unordered_map<int, vector<int>> user_maps_pool;
    user_map(user_maps_pool, queries, sig_maritx, 247753, 50, 5);
    unordered_map<int, vector<int>> user_maps_seed;

    user_map(user_maps_seed, queries, sig_maritx, 33670,50, 5);
    for(int i=0; i<32; i++){
        cout<<user_maps_seed[i].size()<<"  "<< user_maps_pool[i].size()<<endl;
    }
    clock_t time3 = clock();
    cout<<"init sigmatrix time: "<<(time2-time1)/CLOCKS_PER_SECOND<<endl;
    cout<<"init lshmatrix time: "<<(time3-time2)/CLOCKS_PER_SECOND<<endl;

    vector<bucket_info> centroid_angle;
    calculate_centroid_angle(centroid_angle, user_maps_seed, queries, 50, 5);

    priority_queue<canducate_user> top_k;
    priority_queue<uncertain_user> user_pool;

    gen_ExAudiences_cudaOpt(top_k, user_maps_pool, user_maps_seed, user_pool, centroid_angle,  5, 50, 1000, 247753, 33670, data, queries);
    clock_t time4 = clock();

    cout<<"query time: "<<(time4-time3)/CLOCKS_PER_SECOND<<endl;
//    while(!top_k.empty()){
//        cout<<"NO.:"<<top_k.top().sn<<" Sim:"<<top_k.top().sim<<endl;
//        top_k.pop();
//    }

    return 0;


}